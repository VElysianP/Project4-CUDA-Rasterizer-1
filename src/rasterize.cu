#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>



namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut {
		glm::vec4 pos;

		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own
		//My Own *********************************
		 glm::vec3 ndcPos;   //NDC space position 
		 glm::vec2 viewPos; //position in viewport (float)
		 //End ***************************
		 glm::vec3 eyePos;	// eye space position used for shading
		 glm::vec3 eyeNor;	// eye space normal used for shading, cuz normal will go wrong after perspective transformation
		// glm::vec3 col;
		 glm::vec2 texcoord0;
		 TextureData* dev_diffuseTex = NULL;
		// int texWidth, texHeight;
		// ...
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
		glm::vec3 color;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		 glm::vec3 eyePos;	// eye space position used for shading
		 //glm::vec3 eyeNor;
		 glm::vec3 baryCoord;
		 glm::vec2 viewPos;
		// VertexAttributeTexcoord texcoord0;
		// TextureData* dev_diffuseTex;
		// ...
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;


static int width = 0;
static int height = 0;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;

static int * dev_depth = NULL;	// you might need this buffer when doing depth test

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

/** 
* Writes fragment colors to the framebuffer
*/
__global__
void render(int w, int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
		// TODO: add your fragment shader code here
		//Lambert's Cosine Law 
		glm::vec3 color;
		//color = fragmentBuffer[index].color * glm::dot(fragmentBuffer[index].eyePos, fragmentBuffer[index].color);
		color = fragmentBuffer[index].color*abs(glm::dot(fragmentBuffer[index].eyePos, fragmentBuffer[index].color));
		framebuffer[index] = color;
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
	hipFree(dev_fragmentBuffer);
	hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height * sizeof(int));

	checkCUDAError("rasterizeInit");
}

__global__
void initDepth(int w, int h, int * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);
		depth[index] = INT_MAX;
	}
}


/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__ 
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) {
	
	// Attribute (vec3 position)
	// component (3 * float)
	// byte (4 * byte)

	// id of component
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		int count = i / n;
		int offset = i - count * n;	// which component of the attribute

		for (int j = 0; j < componentTypeByteSize; j++) {
			
			dev_dst[count * componentTypeByteSize * n 
				+ offset * componentTypeByteSize 
				+ j]

				= 

			dev_src[byteOffset 
				+ count * (byteStride == 0 ? componentTypeByteSize * n : byteStride) 
				+ offset * componentTypeByteSize 
				+ j];
		}
	}
	

}

__global__
void _nodeMatrixTransform(
	int numVertices,
	VertexAttributePosition* position,
	VertexAttributeNormal* normal,
	glm::mat4 MV, glm::mat3 MV_normal) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
		normal[vid] = glm::normalize(MV_normal * normal[vid]);
	}
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) {
	
	glm::mat4 curMatrix(1.0);

	const std::vector<double> &m = n.matrix;
	if (m.size() > 0) {
		// matrix, copy it

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				curMatrix[i][j] = (float)m.at(4 * i + j);
			}
		}
	} else {
		// no matrix, use rotation, scale, translation

		if (n.translation.size() > 0) {
			curMatrix[3][0] = n.translation[0];
			curMatrix[3][1] = n.translation[1];
			curMatrix[3][2] = n.translation[2];
		}

		if (n.rotation.size() > 0) {
			glm::mat4 R;
			glm::quat q;
			q[0] = n.rotation[0];
			q[1] = n.rotation[1];
			q[2] = n.rotation[2];

			R = glm::mat4_cast(q);
			curMatrix = curMatrix * R;
		}

		if (n.scale.size() > 0) {
			curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
		}
	}

	return curMatrix;
}

void traverseNode (
	std::map<std::string, glm::mat4> & n2m,
	const tinygltf::Scene & scene,
	const std::string & nodeString,
	const glm::mat4 & parentMatrix
	) 
{
	const tinygltf::Node & n = scene.nodes.at(nodeString);
	glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
	n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

	auto it = n.children.begin();
	auto itEnd = n.children.end();

	for (; it != itEnd; ++it) {
		traverseNode(n2m, scene, *it, M);
	}
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) {

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}



	// 2. for each mesh: 
	//		for each primitive: 
	//			build device buffer of indices, materail, and each attributes
	//			and store these pointers in a map
	{

		std::map<std::string, glm::mat4> nodeString2Matrix;
		auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

		{
			auto it = rootNodeNamesList.begin();
			auto itEnd = rootNodeNamesList.end();
			for (; it != itEnd; ++it) {
				traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
			}
		}


		// parse through node to access mesh

		auto itNode = nodeString2Matrix.begin();
		auto itEndNode = nodeString2Matrix.end();
		for (; itNode != itEndNode; ++itNode) {

			const tinygltf::Node & N = scene.nodes.at(itNode->first);
			const glm::mat4 & matrix = itNode->second;
			const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

			auto itMeshName = N.meshes.begin();
			auto itEndMeshName = N.meshes.end();

			for (; itMeshName != itEndMeshName; ++itMeshName) {

				const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

				auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
				std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

				// for each primitive
				for (size_t i = 0; i < mesh.primitives.size(); i++) {
					const tinygltf::Primitive &primitive = mesh.primitives[i];

					if (primitive.indices.empty())
						return;

					// TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
					VertexIndex* dev_indices = NULL;
					VertexAttributePosition* dev_position = NULL;
					VertexAttributeNormal* dev_normal = NULL;
					VertexAttributeTexcoord* dev_texcoord0 = NULL;

					// ----------Indices-------------

					const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
					BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

					// assume type is SCALAR for indices
					int n = 1;
					int numIndices = indexAccessor.count;
					int componentTypeByteSize = sizeof(VertexIndex);
					int byteLength = numIndices * n * componentTypeByteSize;

					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					hipMalloc(&dev_indices, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numIndices,
						(BufferByte*)dev_indices,
						dev_bufferView,
						n,
						indexAccessor.byteStride,
						indexAccessor.byteOffset,
						componentTypeByteSize);


					checkCUDAError("Set Index Buffer");


					// ---------Primitive Info-------

					// Warning: LINE_STRIP is not supported in tinygltfloader
					int numPrimitives;
					PrimitiveType primitiveType;
					switch (primitive.mode) {
					case TINYGLTF_MODE_TRIANGLES:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices / 3;
						break;
					case TINYGLTF_MODE_TRIANGLE_STRIP:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_TRIANGLE_FAN:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_LINE:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices / 2;
						break;
					case TINYGLTF_MODE_LINE_LOOP:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices + 1;
						break;
					case TINYGLTF_MODE_POINTS:
						primitiveType = PrimitiveType::Point;
						numPrimitives = numIndices;
						break;
					default:
						// output error
						break;
					};


					// ----------Attributes-------------

					auto it(primitive.attributes.begin());
					auto itEnd(primitive.attributes.end());

					int numVertices = 0;
					// for each attribute
					for (; it != itEnd; it++) {
						const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
						const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

						int n = 1;
						if (accessor.type == TINYGLTF_TYPE_SCALAR) {
							n = 1;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC2) {
							n = 2;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC3) {
							n = 3;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC4) {
							n = 4;
						}

						BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
						BufferByte ** dev_attribute = NULL;

						numVertices = accessor.count;
						int componentTypeByteSize;

						// Note: since the type of our attribute array (dev_position) is static (float32)
						// We assume the glTF model attribute type are 5126(FLOAT) here

						if (it->first.compare("POSITION") == 0) {
							componentTypeByteSize = sizeof(VertexAttributePosition) / n;
							dev_attribute = (BufferByte**)&dev_position;
						}
						else if (it->first.compare("NORMAL") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
							dev_attribute = (BufferByte**)&dev_normal;
						}
						else if (it->first.compare("TEXCOORD_0") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
							dev_attribute = (BufferByte**)&dev_texcoord0;
						}

						std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

						dim3 numThreadsPerBlock(128);
						dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
						int byteLength = numVertices * n * componentTypeByteSize;
						hipMalloc(dev_attribute, byteLength);

						_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
							n * numVertices,
							*dev_attribute,
							dev_bufferView,
							n,
							accessor.byteStride,
							accessor.byteOffset,
							componentTypeByteSize);

						std::string msg = "Set Attribute Buffer: " + it->first;
						checkCUDAError(msg.c_str());
					}

					// malloc for VertexOut
					VertexOut* dev_vertexOut;
					hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
					checkCUDAError("Malloc VertexOut Buffer");

					// ----------Materials-------------

					// You can only worry about this part once you started to 
					// implement textures for your rasterizer
					TextureData* dev_diffuseTex = NULL;
					int diffuseTexWidth = 0;
					int diffuseTexHeight = 0;
					if (!primitive.material.empty()) {
						const tinygltf::Material &mat = scene.materials.at(primitive.material);
						printf("material.name = %s\n", mat.name.c_str());

						if (mat.values.find("diffuse") != mat.values.end()) {
							std::string diffuseTexName = mat.values.at("diffuse").string_value;
							if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
								const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
								if (scene.images.find(tex.source) != scene.images.end()) {
									const tinygltf::Image &image = scene.images.at(tex.source);

									size_t s = image.image.size() * sizeof(TextureData);
									hipMalloc(&dev_diffuseTex, s);
									hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);
									
									diffuseTexWidth = image.width;
									diffuseTexHeight = image.height;

									checkCUDAError("Set Texture Image data");
								}
							}
						}

						// TODO: write your code for other materails
						// You may have to take a look at tinygltfloader
						// You can also use the above code loading diffuse material as a start point 
					}


					// ---------Node hierarchy transform--------
					hipDeviceSynchronize();
					
					dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					_nodeMatrixTransform << <numBlocksNodeTransform, numThreadsPerBlock >> > (
						numVertices,
						dev_position,
						dev_normal,
						matrix,
						matrixNormal);

					checkCUDAError("Node hierarchy transformation");

					// at the end of the for loop of primitive
					// push dev pointers to map
					primitiveVector.push_back(PrimitiveDevBufPointers{
						primitive.mode,
						primitiveType,
						numPrimitives,
						numIndices,
						numVertices,

						dev_indices,
						dev_position,
						dev_normal,
						dev_texcoord0,

						dev_diffuseTex,
						diffuseTexWidth,
						diffuseTexHeight,

						dev_vertexOut	//VertexOut
					});

					totalNumPrimitives += numPrimitives;

				} // for each primitive

			} // for each mesh

		} // for each node

	}
	

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
	}
	

	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());
			
			//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}


}


//TODO
//have already in NDC space
__device__ void ToClippingSpaceTransform(int index, glm::mat4 MVP,glm::mat4 MV,glm::mat3 MV_normal, PrimitiveDevBufPointers& primitive)
{
	VertexOut tempVertexOut;
	
	tempVertexOut.pos = glm::vec4(primitive.dev_position[index].x, primitive.dev_position[index].y, primitive.dev_position[index].z, 1.f);
	tempVertexOut.eyePos = glm::vec3(MV*tempVertexOut.pos);
	tempVertexOut.eyeNor = glm::normalize(MV_normal*primitive.dev_normal[index]);
	glm::vec4 ClippingPos = MVP*tempVertexOut.pos;
	tempVertexOut.ndcPos = glm::vec3(ClippingPos / ClippingPos.w);

	primitive.dev_verticesOut[index] = tempVertexOut;
}

__global__ 
void _vertexTransformAndAssembly(
	int numVertices, 
	PrimitiveDevBufPointers primitive, 
	glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal, 
	int width, int height) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {

		// TODO: Apply vertex transformation here
		// Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
		// Then divide the pos by its w element to transform into NDC space
		// Finally transform x and y to viewport space

		// TODO: Apply vertex assembly here
		// Assemble all attribute arraies into the primitive array

		ToClippingSpaceTransform(vid, MVP, MV, MV_normal, primitive);

		primitive.dev_verticesOut[vid].viewPos.x = 0.5f * (float)width * (primitive.dev_verticesOut[vid].ndcPos.x + 1.0f);
		primitive.dev_verticesOut[vid].viewPos.y = 0.5f * (float)height * (primitive.dev_verticesOut[vid].ndcPos.y + 1.0f);
	}
}

static int curPrimitiveBeginId = 0;

__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {

		// TODO: uncomment the following code for a start
		// This is primitive assembly for triangles

		int pid;	// id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) {
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
			dev_primitives[pid + curPrimitiveBeginId].primitiveType = Triangle;
		}

		// TODO: other primitive types (point, line)
		//TODO EXTRA POINT 
		if (primitive.primitiveMode == TINYGLTF_MODE_POINTS)
		{
			//the total amount of primitives equals to the amount of vertices and also indices 
			dev_primitives[iid + curPrimitiveBeginId].v[0] = primitive.dev_verticesOut[iid];
			dev_primitives[iid + curPrimitiveBeginId].primitiveType = Point;
			//pid = iid / (int)primitive.primitiveType;
			//dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
			//	= primitive.dev_verticesOut[primitive.dev_indices[iid]];
			//dev_primitives[pid + curPrimitiveBeginId].primitiveType = Point;
		}
		////TODO EXTRA LINE
		if (primitive.primitiveMode == TINYGLTF_MODE_LINE)
		{
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
			dev_primitives[pid + curPrimitiveBeginId].primitiveType = Line;
		}
	}
	
}

__device__ glm::vec3 InterpolateNormal(const  VertexOut vert0, const  VertexOut vert1, const  VertexOut vert2, glm::vec3 barycentricCoord)
{
	glm::vec3 normalColor = vert0.eyeNor * barycentricCoord.x + vert1.eyeNor*barycentricCoord.y + vert2.eyeNor*barycentricCoord.z;
	return normalColor;
}

__device__ glm::vec3 InterpolateEyePos(const  VertexOut vert0, const  VertexOut vert1, const  VertexOut vert2, glm::vec3 barycentricCoord)
{
	glm::vec3 eyePos = vert0.eyePos * barycentricCoord.x + vert1.eyePos*barycentricCoord.y + vert2.eyePos*barycentricCoord.z;
	return eyePos;
}

//TODO trangle rasterization
__device__ void RasterizeTriangle(int w, int h, Fragment *fragmentBuffer, VertexOut vert0, VertexOut vert1, VertexOut vert2, int* depth)
{
	//int pixel indices 
	glm::vec3 vertPixel[3];
	vertPixel[0] = glm::vec3(vert0.viewPos.x, vert0.viewPos.y, 0.f);
	vertPixel[1] = glm::vec3(vert1.viewPos.x, vert1.viewPos.y, 0.f);
	vertPixel[2] = glm::vec3(vert2.viewPos.x, vert2.viewPos.y, 0.f);
	glm::vec3 vertCam[3];
	vertCam[0] = vert0.eyePos;
	vertCam[1] = vert1.eyePos;
	vertCam[2] = vert2.eyePos;

	AABB currentRange = getAABBForTriangle(vertPixel);
	//boundry cases
	if ((currentRange.min.x < 0) || (currentRange.min.y < 0) || (currentRange.max.x >= w) || (currentRange.max.y >= h))
	{
		return;
	}
	for (int j = currentRange.min.y;j <= currentRange.max.y;j++)
	{
		for (int i = currentRange.min.x;i <= currentRange.max.x;i++)
		{
			glm::vec2 testPixel = glm::vec2(i, j);
			int testPixelIndex = testPixel.x + testPixel.y*w;
			glm::vec3 baryCoord = calculateBarycentricCoordinate(vertPixel, testPixel);

			//if it is within the trangle
			if (isBarycentricCoordInBounds(baryCoord))
			{
				float testZDepth = getZAtCoordinate(baryCoord, vertCam);
				//should consider how to one to one map float to int
				int scaledTestZDepth = floor(10000 * testZDepth);

				int lastDepth = atomicMin(&depth[testPixelIndex], scaledTestZDepth);
				if (lastDepth > scaledTestZDepth)
				{
					fragmentBuffer[testPixelIndex].color = InterpolateNormal(vert0,vert1,vert2,baryCoord);
					fragmentBuffer[testPixelIndex].viewPos = testPixel;
					fragmentBuffer[testPixelIndex].baryCoord = baryCoord;
					fragmentBuffer[testPixelIndex].eyePos = InterpolateEyePos(vert0, vert1, vert2, baryCoord);
				}
			}
		}
	}
}

__device__ void RasterizeLine() {

}

__device__ void RasterizePoint(int w, int h, const VertexOut vert0, Fragment* fragmentBuffer, int* depth)
{
	int pointScreenIndex = vert0.viewPos.x + vert0.viewPos.y*w;
	int scaledTestZDepth = floor(10000*vert0.eyePos.z);
	int lastDepth = atomicMin(&depth[pointScreenIndex], scaledTestZDepth);
	if (lastDepth > scaledTestZDepth)
	{
		fragmentBuffer[pointScreenIndex].viewPos = vert0.viewPos;
		fragmentBuffer[pointScreenIndex].color = vert0.eyeNor;
		fragmentBuffer[pointScreenIndex].eyePos = vert0.eyePos;
	}
}

//TODO Rasterization
__global__ void RasterizeGlobal(int numPrimitives,int w, int h, Fragment *fragmentBuffer, Primitive* dev_primitives,int* depth)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index<numPrimitives) {
		Primitive tempPrim = dev_primitives[index];
		if (tempPrim.primitiveType==Triangle)
		{
			VertexOut vert0 = dev_primitives[index].v[0];
			VertexOut vert1 = dev_primitives[index].v[1];
			VertexOut vert2 = dev_primitives[index].v[2];

			RasterizeTriangle(w, h, fragmentBuffer, vert0, vert1, vert2, depth);
		}

		if (tempPrim.primitiveType == Line)
		{
			VertexOut vert0 = dev_primitives[index].v[0];
			VertexOut vert1 = dev_primitives[index].v[1];

			RasterizeLine();
		}

		if (tempPrim.primitiveType == Point)
		{
			VertexOut vert0 = dev_primitives[index].v[0];

			RasterizePoint(w, h, vert0, fragmentBuffer, depth);
		}
	}
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & MVP, const glm::mat4 & MV, const glm::mat3 MV_normal) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	dim3 numThreadsPerBlock(128);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
		curPrimitiveBeginId = 0;
	
		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				//TODO***********************for POINTS*******************
				//p->primitiveMode = TINYGLTF_MODE_POINTS;
				//p->numIndices = p->numVertices;
				//p->numPrimitives = p->numVertices;
				//********************end*********************

				//TODO***********************for LINES (not line loop)*********
				//p->primitiveMode = TINYGLTF_MODE_LINE;
				//p->numPrimitives = p->numVertices - 1;
				//p->numIndices = p->numPrimitives * 2;
				//***********************end**********************8

				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				_vertexTransformAndAssembly << < numBlocksForVertices, numThreadsPerBlock >> >(p->numVertices, *p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				
				_primitiveAssembly << < numBlocksForIndices, numThreadsPerBlock >> >
					(p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");
	}
	
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
	initDepth << <blockCount2d, blockSize2d >> >(width, height, dev_depth);
	
	// TODO: rasterize
	dim3 numBlocksForPrimitives((totalNumPrimitives + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
	RasterizeGlobal << <numBlocksForPrimitives, numThreadsPerBlock >> > (totalNumPrimitives,width,height, dev_fragmentBuffer,dev_primitives,dev_depth);


    // Copy depthbuffer colors into framebuffer
	render << <blockCount2d, blockSize2d >> >(width, height, dev_fragmentBuffer, dev_framebuffer);
	checkCUDAError("fragment shader");
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

    // deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_diffuseTex);

			hipFree(p->dev_verticesOut);

			
			//TODO: release other attributes and materials
		}
	}

	////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

	hipFree(dev_fragmentBuffer);
	dev_fragmentBuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

    checkCUDAError("rasterize Free");
}
